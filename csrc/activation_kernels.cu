#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>

namespace vllm {

// silu_mult
// silu_mult是融合算子，将silu与下一步的乘加运算融合到一起进行计算。
// silu函数很容易理解，就是按照silu公式写的函数。 怀疑silu_mult中input的2d是相同的值，
// 那么对应的公式便是 y = x * silu(x)
// ldg的作用：ldg会将数据从全局内存中搬运到blcok内的纹理缓存中。
template<typename T>
__device__ __forceinline__ T silu(const T& x) {
  // x * sigmoid(x)
  return (T) (((float) x) / (1.0f + expf((float) -x)));
}

template<typename scalar_t>
__global__ void silu_and_mul_kernel(
  scalar_t* __restrict__ out,               // [num_tokens, d]
  const scalar_t* __restrict__ input,       // [num_tokens, 2, d]
  const int d) {
  const int token_idx = blockIdx.x;
  for (int idx = threadIdx.x; idx < d; idx += blockDim.x) {
    const scalar_t x = __ldg(&input[token_idx * 2 * d + idx]);
    const scalar_t y = __ldg(&input[token_idx * 2 * d + d + idx]);
    out[token_idx * d + idx] = silu(x) * y;
  }
}

} // namespace vllm

void silu_and_mul(
  torch::Tensor& out,      // [num_tokens, d]
  torch::Tensor& input)    // [num_tokens, 2 * d]
{
  int num_tokens = input.size(0);
  int d = input.size(1) / 2;

  dim3 grid(num_tokens);
  dim3 block(std::min(d, 1024));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES_AND2(
    at::ScalarType::Half,
    at::ScalarType::BFloat16,
    input.scalar_type(),
    "silu_and_mul_kernel",
    [&] {
      vllm::silu_and_mul_kernel<scalar_t><<<grid, block, 0, stream>>>(
        out.data_ptr<scalar_t>(),
        input.data_ptr<scalar_t>(),
        d);
    });
}
